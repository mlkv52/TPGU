
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float s=0;

__global__ void gInit(float* a, float* b){
 int i=threadIdx.x+blockIdx.x*blockDim.x;

 a[i]=0.01*i;
 b[i]=0.01*1.0;
}
__global__ void gScalarProduct(float* a, float* b){
 int i=threadIdx.x+blockIdx.x*blockDim.x;
/*
 __shared__ float portion[1024];

 portion[threadIdx.x]=a[i]*b[i];
 __syncthreads();

 for (int n = blockDim.x / 2; n > 0; n >>= 1){
    if (threadIdx.x < n)
      portion[threadIdx.x] += portion[threadIdx.x + n];
    __syncthreads();
  } 
  __syncthreads();
  if(threadIdx.x==0){
     atomicAdd(&s, portion[0]);
   }
   */
   atomicAdd(&s, a[i]*b[i]);
}

int main(){
 float *a, *b;

 int N=1<<20;
 hipMalloc((void**)&a, N*sizeof(float));
 hipMalloc((void**)&b, N*sizeof(float));

 gInit<<<N/1024,1024>>>(a,b);
 hipDeviceSynchronize();
 
 gScalarProduct<<<N/1024,1024>>>(a,b);
 hipDeviceSynchronize();

 float s_h;
 hipMemcpyFromSymbol(&s_h, HIP_SYMBOL(s), sizeof(float));
 printf("%g\n", s_h);

 return 0;
}
